#include "hip/hip_runtime.h"
#include "gamma_op.h"
#include <cmath>

namespace Utils {
namespace Ops {


GammaOp::GammaOp() {
  _gamma = 2.2;
}


void GammaOp::apply_cpu(const float* src, float* dst,
                     size_t H, size_t W, size_t C) {
  #pragma omp parallel for
  for (size_t i = 0; i < H * W * C; ++i)
    dst[i] = pow(src[i], _gamma);

}

void GammaOp::apply_gpu(const float* src, float* dst,
                     size_t H, size_t W, size_t C) {
  // TODO
  apply_cpu(src, dst, H, W, C);
}

}
}
