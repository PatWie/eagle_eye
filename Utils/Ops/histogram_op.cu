#include "hip/hip_runtime.h"
#include "histogram_op.h"
#include <cmath>

#include <glog/logging.h>

namespace Utils {
namespace Ops {
#ifdef CUDA_ENABLED
namespace {
__global__ void cuda(const float *src, float *dst,
                     size_t H, size_t W, size_t C,
                     Utils::Ops::HistogramOp::scaling_t _scaling) {
  const int c = blockIdx.z;
  for (int w = blockIdx.x * blockDim.x + threadIdx.x; w < W; w += blockDim.x * gridDim.x) {
    for (int h = blockIdx.y * blockDim.y + threadIdx.y; h < H; h += blockDim.y * gridDim.y) {
      float inp = src[c * H * W + h * W + w];
      float scaled = inp / _scaling.scale;
      scaled -= _scaling.min / _scaling.scale;
      scaled /= (_scaling.max - _scaling.min) / _scaling.scale;
      // apply clipping
      scaled = min(scaled, 1.f);
      scaled = max(scaled, 0.f);
      dst[c * H * W + h * W + w] = scaled;
    }
  }
}
}; // anonymous namespace
#endif // CUDA_ENABLED

HistogramOp::HistogramOp() {}

void HistogramOp::apply_cpu(const float* src, float* dst,
                            size_t H, size_t W, size_t C) {

  #pragma omp parallel for
  for (size_t i = 0; i < H * W * C; ++i) {
    dst[i] = src[i];
    // map from [0, img_max] _> [0, 1]
    float scaled = src[i] / _scaling.scale;
    scaled -= _scaling.min / _scaling.scale;
    scaled /= (_scaling.max - _scaling.min) / _scaling.scale;
    // apply clipping
    scaled = std::min(scaled, 1.f);
    dst[i] = std::max(scaled, 0.f);
  }
}

#ifdef CUDA_ENABLED
void HistogramOp::apply_gpu(const float* src, float* dst,
                            size_t H, size_t W, size_t C) {

  float *d_src;
  float *d_dst;

  hipMalloc(&d_src, sizeof(float) * H * W * C);
  hipMalloc(&d_dst, sizeof(float) * H * W * C);

  hipMemcpy(d_src, src, sizeof(float) * H * W * C, hipMemcpyHostToDevice);
  hipMemcpy(d_dst, dst, sizeof(float) * H * W * C, hipMemcpyHostToDevice);

  const int num_threads = 32;
  dim3 threads(num_threads, num_threads);
  dim3 grid((W - 1) / threads.x + 1, (H - 1) / threads.y + 1, C);
  cuda <<< grid, threads>>> (d_src, d_dst, H, W, C, _scaling);
  hipDeviceSynchronize();

  hipMemcpy(dst, d_dst, sizeof(float) * H * W * C, hipMemcpyDeviceToHost);

  hipFree(d_src);
  hipFree(d_dst);

}
#endif // CUDA_ENABLED


}
}
